
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
using namespace std;

//matrix structure
typedef struct {
	int n;
	int *el;
} Matrix;


//KERNEL
// D = AxB
__global__ void calcD(int n, Matrix D, Matrix A, Matrix B, Matrix C)
{
	int Cv = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e=0; e<n; ++e)
		Cv+=A.el[row*n+e] * B.el[e*n+col];	//calculate one value
	D.el[row*n+col]=Cv+C.el[row*n+col];		//add value from C and write to D
}

//HOST CODE
int main(int argc, char* argv[])
{
	if (argc<2) {
		cout << "Input file not specified. Please, specify it as a first argument." << endl;
		cout << "example: " << argv[0] << " matr.txt" << endl;
		return -1;
	} 	
	ifstream file(argv[1]);
	ofstream output("output.txt");

	if (!file)
	{
		cout << "Error opening file" << endl;
		return -1;
	}

	int n;
	file >> n;			//size N
	if (argc>2) cout << "N=" << n << endl;

	Matrix A, B, C, D;	//host matrices
	A.el = new int[n*n];	//....	
	B.el = new int[n*n];	//...
	C.el = new int[n*n];	//..
	D.el = new int[n*n];	//.

	//reading from file into matrices
	for (int i=0; i<(n*n); i++)
		file >> A.el[i];
	for (int i=0; i<n*n; i++)
		file >> B.el[i];
	for (int i=0; i<n*n; i++)
		file >> C.el[i];

	//preparing for the device
	Matrix d_A;
	d_A.n=n;
	size_t size = n*n*sizeof(int);
	hipMalloc(&d_A.el, size);		//allocate memory for A
	hipMemcpy(d_A.el, A.el, size, hipMemcpyHostToDevice);	//copy A to deviceA (d_A)
	
	Matrix d_B;
	d_B.n=n;
	hipMalloc(&d_B.el, size);		//same for B
	hipMemcpy(d_B.el, B.el, size, hipMemcpyHostToDevice);

	Matrix d_C;
	d_C.n=n;
	hipMalloc(&d_C.el, size);		//same for C
	hipMemcpy(d_C.el, C.el, size, hipMemcpyHostToDevice);

	Matrix d_D;				//resulting matrix D
	d_D.n=n;
	size = n*n*sizeof(int);
	hipMalloc(&d_D.el, size);		//only allocate memory
	
	//kernel call
	dim3 dimBlock(n,n);	// USING ONE BLOCK
	dim3 dimGrid(1,1);	// WITH NxN THREADS
	calcD<<<dimGrid, dimBlock>>>(n, d_D, d_A, d_B, d_C);

	//read matrix E back
	hipMemcpy(D.el, d_D.el, size, hipMemcpyDeviceToHost);

	//write output to file
	output << "Matrix D:" << endl;
	for (int i=0; i<n; i++)	{
		for (int j=0; j<n; j++)	
			output << D.el[(i*n+j)] << " ";
		output << endl;
	}
	
	//print out resulting matrix D if second argument was present
	if (argc>2) {
		cout << endl << "Matrix D:" << endl;
		for (int i=0; i<n; i++)	{
			for (int j=0; j<n; j++)	
				cout << D.el[(i*n+j)] << " ";
			cout << endl;
		}
	}
	
	//free the memory on device
	hipFree(d_A.el);
	hipFree(d_B.el);
	hipFree(d_C.el);
	hipFree(d_D.el);
	//free the memory on host
	delete[] A.el;
	delete[] B.el;
	delete[] C.el;
	delete[] D.el;
	file.close();
	output.close();
	cout << endl << "Done. " << endl;
	return 0;
}